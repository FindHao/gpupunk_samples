#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Host main routine
 */
int main(void)
{
    // Print the vector length to be used, and compute its size
    int numElements = 1;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);
    float *A, *B, *C;
    checkCudaErrors(hipMallocManaged(&A, numElements * sizeof(float)));
    checkCudaErrors(hipMallocManaged(&B, numElements * sizeof(float)));
    checkCudaErrors(hipMallocManaged(&C, numElements * sizeof(float)));

    A[0] = 0.123456789;
    B[0] = 0.1;

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, numElements);
    hipDeviceSynchronize();

    printf("%f\n", C[0]);
    if (abs(C[0] - 0.2234567) < 1e-6)
        printf("Test PASSED\n");
    else{
        printf("Test Failed.\n");
    }

    // Free device global memory
    checkCudaErrors(hipFree(A));
    checkCudaErrors(hipFree(B));
    checkCudaErrors(hipFree(C));
    printf("Done\n");
    return 0;
}
